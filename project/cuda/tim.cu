#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   tim.cu
**   Author: taparson
**   Created: 8 Apr 2014
**
**************************************************************************/

#ifndef TIM_CU
#define TIM_CU

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#ifndef GLM_FORCE_RADIANS
    #define GLM_FORCE_RADIANS
#endif
#include <glm/geometric.hpp>
#include <glm/vec3.hpp>
#include <glm/mat3x3.hpp>

#define CUDA_INCLUDE
#include "sim/particle.h"
#include "geometry/grid.h"
#include "cuda/functions.h"

extern "C"  {
void groupParticlesTests();
void cumulativeSumTests();
void CSTest1();
void CSTest2();
void CSTest3();
void CSTest4();
void CSTest5();
void PGTest1();

}



__host__ __device__ void gridIndexToIJK(int idx, int &i, int &j, int &k,const  glm::ivec3 &nodeDim){
    i = idx / (nodeDim.y*nodeDim.z);
    idx = idx % (nodeDim.y*nodeDim.z);
    j = idx / nodeDim.z;
    k = idx % nodeDim.z;
}

__host__ __device__ int getGridIndex( int i, int j, int k, const glm::ivec3 &nodeDim)  {
    return (i*(nodeDim.y*nodeDim.z) + j*(nodeDim.z) + k);
}

__host__ __device__ void gridIndexToIJK(int idx, const  glm::ivec3 &nodeDim, glm::ivec3 &IJK){
    gridIndexToIJK(idx, IJK.x, IJK.y, IJK.z, nodeDim);
}

__host__ __device__ int getGridIndex( const glm::ivec3 &IJK, const glm::ivec3 &nodeDim)  {
    return getGridIndex(IJK.x, IJK.y, IJK.z, nodeDim);
}



__host__ __device__ void positionToGridIJK(vec3 pos, Grid *grid, int &i, int &j, int &k){
    pos-=grid->pos;
    pos/=grid->h;
    pos = vec3::round(pos);
    i = (int) pos.x;
    j = (int) pos.y;
    k = (int) pos.z;
}

__host__ __device__ void positionToGridIJK(vec3 &pos, Grid *grid, glm::ivec3 &gridIJK){
    pos-=grid->pos;
    pos /= grid->h;
    pos = vec3::round(pos);
    gridIJK = glm::ivec3((int) pos.x, (int) pos.y, (int) pos.z);
}

/**
* Assuming N = # particles, M = dim.x*dim.y*dim.z for grid.
* naming convention: things that start with “particle” have N items, things that start with “cell” have M items.
* particleData: Array of type Particle, simply a list of all of our particles, size N
* grid: Grid dimensions and unit size
* particleToCell: Array of type int, size N, index of cell that particle belongs to.
* cellParticleCount: Array of type int, size M, number of particles in each cell.
* particleOffsetInCell: Array of type int, size N, offset for each particle into cell’s subarray. (number of particles already inserted into the cell that the particle belongs to)
*
*/
__global__ void rasterizeParticles( Particle *particleData, Grid *grid, int *particleToCell, int *cellParticleCount, int *particleOffsetInCell ) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    Particle p = particleData[index];
    glm::ivec3 gridIJK;
    //positionToGridIJK(p.position, grid, gridIJK);
    int gridIndex = getGridIndex(gridIJK.x, gridIJK.y, gridIJK.z, grid->dim+1);
    particleToCell[index] = gridIndex;
    particleOffsetInCell[index]=cellParticleCount[gridIndex]++;
}

__global__ void cumulativeSum(int *array, int M)  {
    int sum = 0;
    for(int i = 0; i < M; i++)  {
        sum+=array[i];
        array[i] = sum;
    }
}

/**
 * particleToCell: Array of type int, size N, index of the cell that particle belongs to.
 * cellParticleIndex: Array of type int, size M, index of first particle for each cell
 * particleOffsetInCell: Array of type int, size N, offset for each particle into cell’s subarray
 * gridParticles: Array of type int, size N, particle indices group by ascending cell index
 */
__global__ void groupParticlesByCell( int *particleToCell, int *cellParticleIndex, int *particleOffsetInCell, int *gridParticles )  {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int gridIndex = particleToCell[index];
    int subPosition = particleOffsetInCell[index];
    int resultIndex = cellParticleIndex[gridIndex] + subPosition;
    gridParticles[resultIndex] = index;
}

void groupParticlesTests()  {
    printf("running particle grouping tests...\n");

    PGTest1();

    printf("done running particle grouping tests\n");
}

void PGTest1()  {
    int particleToCell[8] = {2,3,2,1,0,7,6,5};
    int cellParticleIndex[9] = {0,1,1,2,1,0,1,1,1};
    int particleOffsetInCell[8] = {0,0,1,0,0,0,0,0};
    int gridParticles[8] = {0,0,0,0,0,0,0,0};
    int *dev_particleToCell, *dev_cellParticleIndex, *dev_particleOffsetInCell, *dev_gridParticles;
    checkCudaErrors(hipMalloc((void**) &dev_particleToCell, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_particleToCell,particleToCell,8*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_cellParticleIndex, 9*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_cellParticleIndex,cellParticleIndex,9*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_particleOffsetInCell, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_particleOffsetInCell,particleOffsetInCell,8*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_gridParticles, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_gridParticles,gridParticles,8*sizeof(int),hipMemcpyHostToDevice));

    cumulativeSum<<<1,1>>>(dev_cellParticleIndex,9);
    hipDeviceSynchronize();
    groupParticlesByCell<<<8,1>>>(dev_particleToCell,dev_cellParticleIndex,dev_particleOffsetInCell,dev_gridParticles);

    hipDeviceSynchronize();
    hipMemcpy(gridParticles,dev_gridParticles,8*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(dev_particleToCell);
    hipFree(dev_cellParticleIndex);
    hipFree(dev_particleOffsetInCell);
    hipFree(dev_gridParticles);
    printf("{");
    for (int i=0; i < 8; i++)  {
        printf("%d,",gridParticles[i]);
    }
    printf("}\n");
}

void cumulativeSumTests()
{
    printf("running cumulative sum tests...\n");
    CSTest1();
    CSTest2();
    CSTest3();
    CSTest4();
    CSTest5();
    printf("done running cumulative sum tests\n");
}

void CSTest1()  {
    int array[5] = {0,1,2,3,4};
    int expected[5] = {0,1,3,6,10};
    printf("running test on array: [%d,%d,%d,%d,%d]...\n",array[0],array[1],array[2],array[3],array[4]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 5*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,5*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,5);
    hipMemcpy(array,dev_array,5*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 5; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d,%d,%d,%d,%d}",expected[0],expected[1],expected[2],expected[3],expected[4]);
            printf("    got: {%d,%d,%d,%d,%d}\n",array[0],array[1],array[2],array[3],array[4]);
            break;
        }
    }
}

void CSTest2()  {
    int array[5] = {5,1,2,3,4};
    int expected[5] = {5,6,8,11,15};
    printf("running test on array: [%d,%d,%d,%d,%d]...\n",array[0],array[1],array[2],array[3],array[4]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 5*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,5*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,5);
    hipMemcpy(array,dev_array,5*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 5; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d,%d,%d,%d,%d}",expected[0],expected[1],expected[2],expected[3],expected[4]);
            printf("    got: {%d,%d,%d,%d,%d}\n",array[0],array[1],array[2],array[3],array[4]);
            break;
        }
    }
}

void CSTest3()  {
    int array[1] = {5};
    int expected[1] = {5};
    printf("running test on array: [%d]...\n",array[0]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 1*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,1*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,1);
    hipMemcpy(array,dev_array,1*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 1; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d}",expected[0]);
            printf("    got: {%d}\n",array[0]);
            break;
        }
    }
}

void CSTest4()  {
    int array[1] = {0};
    int expected[1] = {0};
    printf("running test on array: [%d]...\n",array[0]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 1*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,1*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,1);
    hipMemcpy(array,dev_array,1*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 1; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d}",expected[0]);
            printf("    got: {%d}\n",array[0]);
            break;
        }
    }
}

void CSTest5()  {
    int array[0] = {};
    int expected[0] = {};
    printf("running test on array: []...\n");
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 0*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,0*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,0);
    hipMemcpy(array,dev_array,0*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 0; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {}",expected[0]);
            printf("    got: {}\n");
            break;
        }
    }
}

#endif // TIM_CU

