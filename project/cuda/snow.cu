#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   snow.cu
**   Author: mliberma
**   Created: 7 Apr 2014
**
**************************************************************************/

#ifndef SNOW_CU
#define SNOW_CU

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#ifndef GLM_FORCE_RADIANS
    #define GLM_FORCE_RADIANS
#endif
#include <glm/geometric.hpp>

#define CUDA_INCLUDE
#include "sim/particle.h"
#include "cuda/functions.h"

void registerVBO( hipGraphicsResource **resource, GLuint vbo )
{
    checkCudaErrors( hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsNone) );
}

void unregisterVBO( hipGraphicsResource *resource )
{
    checkCudaErrors( hipGraphicsUnregisterResource(resource) );
}

//__global__ void snow_kernel( float time, Particle *particles )
//{
//    int index = blockIdx.x*blockDim.x + threadIdx.x;
//    vec3 pn = vec3::normalize( particles[index].position );
//    particles[index].position += 0.05f*sinf(6*time)*pn;
////    particles[index].position += 0.01f*pn;
//}

//void updateParticles( Particle *particles, float time, int particleCount )
//{
//    snow_kernel<<< particleCount/512, 512 >>>( time, particles );
//    checkCudaErrors( hipDeviceSynchronize() );
//}

#endif // SNOW_CU
